
#include "gf_matrix.h"
#include "cuder.h"

constexpr bool PRINT_VERBOSE = 0;
constexpr bool PRINT_INITIAL_VALUE = 0;
constexpr bool PRINT_RESULT = 0;



constexpr int M = 2048;

constexpr int BITS = 8;

using gf_int_t = gf_int<BITS>;
using square_t = gf_square<M, BITS>;



square_t h_A;
square_t h_B;

int main(void)
{
	cuder<square_t> d_A_ptr(make_cuder<square_t>());
	cuder<square_t> d_B_ptr(make_cuder<square_t>());
	cuder<gf_int_t> d_coeff_ptr(make_cuder<gf_int_t>(M));

	for (uint32_t i = 0; i < M; ++i) {
		for (uint32_t j = i; j < M; ++j) {
			h_A.data[i][j] = gf_int_t(i + j + 1);
		}

		h_B.data[i][i] = gf_int_t(1);
	}

	if (PRINT_INITIAL_VALUE) {
		for (uint32_t i = 0; i < M; ++i) {
			for (uint32_t j = 0; j < M; ++j) {
				std::cout << std::hex << h_A.data[i][j] << ' ';
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}
	
	hipMemcpy(d_A_ptr.toKernel(), &h_A, sizeof(h_A), hipMemcpyHostToDevice);
	hipMemcpy(d_B_ptr.toKernel(), &h_B, sizeof(h_B), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	if (PRINT_VERBOSE) {
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
	}

	for (int num_pivot = 0; num_pivot < M; ++num_pivot) {
		Calcu_Row_Coeffs<<<dim3(16, 16), dim3(16, 16)>>>
			(d_A_ptr.toKernel(), d_coeff_ptr.toKernel(), num_pivot);
		hipDeviceSynchronize();

		Eliminate_Rows<<<dim3(16, 16), dim3(16, 16)>>>
			(d_A_ptr.toKernel(), d_B_ptr.toKernel(), d_coeff_ptr.toKernel(), num_pivot);
		hipDeviceSynchronize();

		if (PRINT_VERBOSE) {
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			float elapsedTime;
			hipEventElapsedTime(&elapsedTime, start, stop);
			std::cout << "Round " << num_pivot << ": " << (elapsedTime/1000) << " s" << std::endl;
		}	
	}

	Normalize_By_Pivots<<<dim3(16, 16), dim3(16, 16)>>>
		(d_A_ptr.toKernel(), d_B_ptr.toKernel());
	hipDeviceSynchronize();
	
	hipMemcpy(&h_A, d_A_ptr.toKernel(), sizeof(h_A), hipMemcpyDeviceToHost);
	hipMemcpy(&h_B, d_B_ptr.toKernel(), sizeof(h_B), hipMemcpyDeviceToHost);

	if (PRINT_RESULT) {
		for (uint32_t i = 0; i < M; ++i) {
			for (uint32_t j = 0; j < M; ++j) {
				std::cout << std::hex << h_B.data[i][j] << ' ';
			}
			std::cout << std::endl;
		}
	}

	return 0;
}