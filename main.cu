
#include "gf_matrix.h"
#include "cuder.h"

constexpr int M = 1024;
constexpr int N = M*2;

using gf_int_t = gf_int<CURRENT_BITS>;
using matrix_t = gf_matrix<CURRENT_BITS, M, N>;



matrix_t h_mat;

int main(void)
{
	cuder<matrix_t> d_mat_ptr(make_cuder<matrix_t>());
	cuder<gf_int_t> d_coeff_ptr(make_cuder<gf_int_t>(M));

	for (uint32_t i = 0; i < M; ++i) {
		for (uint32_t j = i; j < M; ++j) {
			h_mat.data[i][j].assigned(matrix_t::data_t(i + j + 1));
		}

		h_mat.data[i][i+M].assigned(matrix_t::data_t(1));
	}
	/*for (uint32_t i = 0; i < M; ++i) {
		for (uint32_t j = 0; j < M; ++j) {
			std::cout << std::hex << h_mat.data[i][j] << ' ';
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;*/
	
	hipMemcpy(d_mat_ptr.toKernel(), &h_mat, sizeof(h_mat), hipMemcpyHostToDevice);

	Inverse_Precheck<<<dim3(16, 16), dim3(BLOCK_DIM_X, BLOCK_DIM_Y, CURRENT_BITS)>>>
		(d_mat_ptr.toKernel());
	hipDeviceSynchronize();

	for (int num_pivot = 0; num_pivot < M; ++num_pivot) {
		Calcu_Row_Coeffs<<<dim3(16), dim3(BLOCK_DIM_X, 1, CURRENT_BITS)>>>
			(d_mat_ptr.toKernel(), d_coeff_ptr.toKernel(), num_pivot);
		hipDeviceSynchronize();

		Eliminate_Rows<<<dim3(16, 16), dim3(BLOCK_DIM_X, BLOCK_DIM_Y, CURRENT_BITS)>>>
			(d_mat_ptr.toKernel(), d_coeff_ptr.toKernel(), num_pivot);
		hipDeviceSynchronize();
	}

	Normalize_By_Pivots<<<dim3(16, 16), dim3(BLOCK_DIM_X, BLOCK_DIM_Y, CURRENT_BITS)>>>
		(d_mat_ptr.toKernel());
	hipDeviceSynchronize();
	
	hipMemcpy(&h_mat, d_mat_ptr.toKernel(), sizeof(h_mat), hipMemcpyDeviceToHost);
	/*for (uint32_t i = 0; i < M; ++i) {
		for (uint32_t j = M; j < N; ++j) {
			std::cout << std::hex << h_mat.data[i][j] << ' ';
		}
		std::cout << std::endl;
	}*/

	return 0;
}