
#include "gf_matrix.h"
#include "cuder.h"

constexpr bool PRINT_VERBOSE = 0;
constexpr bool PRINT_INITIAL_VALUE = 0;
constexpr bool PRINT_RESULT = 1;



constexpr int M = 32;

constexpr int BITS = 8;

using gf_int_t = gf_int<BITS>;
using square_t = gf_square<gf_int_t, M>;



// Allocate static memory on host
square_t h_A;
square_t h_B;

int main(void)
{
	try {
		// Allocate heap memory on device
		cuder<square_t> d_A_ptr(make_cuder<square_t>());
		cuder<square_t> d_B_ptr(make_cuder<square_t>());

		// Initialize host data and/or print it
		for (uint32_t i = 0; i < M; ++i) {
			for (uint32_t j = i; j < M; ++j) {
				h_A.data[i][j] = gf_int_t(i + j + 1);
			}

			h_B.data[i][i] = gf_int_t(1);
		}

		if (PRINT_INITIAL_VALUE) {
			for (uint32_t i = 0; i < M; ++i) {
				for (uint32_t j = 0; j < M; ++j) {
					std::cout << std::hex << h_A.data[i][j] << ' ';
				}
				std::cout << std::endl;
			}
			std::cout << std::endl;
		}
		
		// Initialize device data
		hipMemcpy(d_A_ptr.toKernel(), &h_A, sizeof(h_A), hipMemcpyHostToDevice);
		hipMemcpy(d_B_ptr.toKernel(), &h_B, sizeof(h_B), hipMemcpyHostToDevice);

		hipEvent_t start, stop;
		if (PRINT_VERBOSE) {
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start, 0);
		}

		dim3 grid(1, 1, 1);
		dim3 block(BLOCK_DIM, BLOCK_DIM);
		shared_op_test<<<grid, block>>>(d_A_ptr.toKernel(), d_B_ptr.toKernel());
		/*for (int num_pivot = 0; num_pivot < M; ++num_pivot) {
			
			hipDeviceSynchronize();

			if (PRINT_VERBOSE) {
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				float elapsedTime;
				hipEventElapsedTime(&elapsedTime, start, stop);
				std::cout << "Round " << num_pivot << ": " << (elapsedTime/1000) << " s" << std::endl;
			}	
		}*/
		
		// Copy the result back to host and/or print it
		hipMemcpy(&h_B, d_B_ptr.toKernel(), sizeof(h_B), hipMemcpyDeviceToHost);

		if (PRINT_RESULT) {
			for (uint32_t i = 0; i < M; ++i) {
				for (uint32_t j = 0; j < M; ++j) {
					std::cout << std::hex << h_B.data[i][j] << ' ';
				}
				std::cout << std::endl;
			}
		}
	
	} catch (std::bad_alloc &e) {
		std::cout << "Failed to allocate enough memory on GPU." << std::endl;
	}



	return 0;
}